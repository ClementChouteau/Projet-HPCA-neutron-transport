#include "hip/hip_runtime.h"
#include "neutron_gpu_kernel.h"

#include <hip/hip_runtime.h>

__global__
void neutron_seq_kernel(long n,
												int neutronsPerThread,
												const ProblemParameters* params,
												float* absorbed,
												long* d_r,
												long* d_b,
												long* d_t,
												unsigned long long* seeds,
												hiprandState* states) {
	const long id = blockIdx.x*blockDim.x + threadIdx.x;

	hiprand_init(seeds[id], id, 0, states+id);

	const float c = params->c;
	const float c_c = params->c_c;
	const float h = params->h;
	long r = 0, b = 0, t = 0;

	long cpt = (blockIdx.x*blockDim.x)*neutronsPerThread + threadIdx.x;
	for (long i=0; i<neutronsPerThread; i++) {
		if (!(cpt < n))
			break;

		float d = 0.0;
		float x = 0.0;

		float v;
		while (1) {

			const float u = hiprand_uniform (states+id);
			float L = -(1 / c) * log(u);
			x = x + L * cos(d);

			if (x < 0) {
				r++;
				v = NO_VAL;
				break;
			}
			else if (x >= h) {
				t++;
				v = NO_VAL;
				break;
			}
			else if (hiprand_uniform (states+id) < c_c / c) {
				b++;
				v = x;
				break;
			}
			else {
				const float u = hiprand_uniform (states+id);
				d = u * M_PI;
			}
		}
		absorbed[cpt+=blockDim.x] = v;
	}

	d_r[id] = r;
	d_b[id] = b;
	d_t[id] = t;
}
