/*
 * Université Pierre et Marie Curie
 * Calcul de transport de neutrons
 * Version séquentielle
 */

#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

#include "neutron.h"

#include <utility>
#include <chrono>
#include <iostream>

#include "neutron_gpu_kernel.h"

using namespace std::chrono;

/**
 * Retourne le quotient entier superieur ou egal a "a/b".
 */
template<typename T>
inline static T iDivUp(T a, T b){
	return ((a % b != 0) ? (a / b + 1) : (a / b));
}

ExperimentalResults neutron_gpu_caller(float* absorbed, long n,
																			 const ProblemParameters& params,
																			 const std::vector<unsigned long long>& seeds,
																			 int threadsPerBlock, int neutronsPerThread) {
	const auto threads = threadsPerBlock*iDivUp<long>(n, threadsPerBlock*neutronsPerThread);

	auto t1 = system_clock::now();
	unsigned long long* d_seeds;
	hipMalloc((void**)&d_seeds, seeds.size()*sizeof(unsigned long long));
	hipMemcpy(d_seeds, seeds.data(), seeds.size()*sizeof(unsigned long long), hipMemcpyHostToDevice);

	// launching cuda kernel
	ProblemParameters* d_params;
	hipMalloc((void**)&d_params, sizeof(ProblemParameters));
	hipMemcpy(d_params, &params, sizeof(ProblemParameters), hipMemcpyHostToDevice);

	unsigned long long int* d_next_absorbed;
	hipMalloc((void**)&d_next_absorbed, sizeof(unsigned long long int));
	hipMemset(d_next_absorbed, 0, sizeof(unsigned long long int));

	float* d_absorbed;
	hipMalloc((void**)&d_absorbed, n*sizeof(float));

	unsigned long long int* d_r, * d_b, * d_t;
	hipMalloc((void**)&d_r, sizeof(unsigned long long int));
	hipMalloc((void**)&d_b, sizeof(unsigned long long int));
	hipMalloc((void**)&d_t, sizeof(unsigned long long int));
	hipMemset(d_r, 0, sizeof(unsigned long long int));
	hipMemset(d_b, 0, sizeof(unsigned long long int));
	hipMemset(d_t, 0, sizeof(unsigned long long int));

	hiprandState* d_states;
	hipMalloc((void**)&d_states, threads*sizeof(hiprandState));
	auto t2 = system_clock::now();
	std::cout << "Temps de la copie CPU -> GPU: " << std::chrono::duration_cast<milliseconds>(t2 - t1).count()/1000. << " sec" << std::endl;

	const dim3 nthreads(threadsPerBlock);
	const dim3 nblocs(iDivUp<long>(n, threadsPerBlock*neutronsPerThread));
	std::cout << "Nombre de blocs GPU: " << nblocs.x << std::endl;
	std::cout << "Nombre de threads par bloc: " << nthreads.x << std::endl;
	std::cout << "Mémoire utilisée: " << (n*4.)/(1024.*1024.) << "Mo" << std::endl;

	auto t3 = system_clock::now();
	neutron_gpu_kernel<<<nthreads, nblocs>>>(n, neutronsPerThread, d_params,
																					 d_next_absorbed, d_absorbed,
																					 d_r, d_b, d_t, d_seeds, d_states);

	// retrieving results
	hipDeviceSynchronize();
	auto t4 = system_clock::now();
	std::cout << "Temps du kernel: " << std::chrono::duration_cast<milliseconds>(t4 - t3).count()/1000. << " sec" << std::endl;

	hipFree(d_next_absorbed),
	hipFree(d_seeds);

	ExperimentalResults res;
	hipMemcpy(&res.r, d_r, sizeof(unsigned long long int), hipMemcpyDeviceToHost);
	hipMemcpy(&res.b, d_b, sizeof(unsigned long long int), hipMemcpyDeviceToHost);
	hipMemcpy(&res.t, d_t, sizeof(unsigned long long int), hipMemcpyDeviceToHost);
	hipFree(d_r);
	hipFree(d_b);
	hipFree(d_t);

	if (res.r+res.b+res.t != n)
		exit(1);

	t1 = system_clock::now();
	res.absorbed = absorbed;

	hipMemcpy(res.absorbed, d_absorbed, res.b*sizeof(float), hipMemcpyDeviceToHost);
	hipFree(d_absorbed);

	t2 = system_clock::now();
	std::cout << "Temps de la copie GPU -> CPU: " << std::chrono::duration_cast<milliseconds>(t2 - t1).count()/1000. << " sec" << std::endl;

	return res;
}
